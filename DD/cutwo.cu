
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>



extern "C" int cucaldog(float * date,int xar,int yar,int xm,int ym);

__global__ void calkel(float * date,int xr,int yr) 
{
    int i,j;
    i=threadIdx.x;
    j=blockIdx.x;
    // printf("%d-%d :%f\n",i,j, date[j*yr+i*xr]);
    date[j*yr+i*xr]=(float)(i+j)*(i-j);
    
}
int cucaldog(float * date,int xar,int yar,int xm,int ym)
{
    int size;
    float * num=NULL;
    xar=xar/sizeof(float);
    yar=yar/sizeof(float);
    size=(ym*yar)*sizeof(float);
    // printf("%f \n",date[(xm-1)*yar+(ym-1)*xar-1]);
    // for(int i = 0; i < ym; i++)
    // {
    //     for (int j = 0; j < xm; j++)
    //     {
    //         printf("%f@%d ",date[j*xar+i*yar],j*xar+i*yar);
    //         // p[j*xar+i*yar]=(float)sin(i+j);
    //     }
    //     printf("\n");
    // }
    hipMallocManaged((void**)&num, size);
    // cudaMemcpy(num, date, size, cudaMemcpyHostToDevice);
    // printf("***%d \n",xar*yar);
    memcpy(num, date,size);
    calkel<<<ym,xm>>>(num,xar,yar);
    hipDeviceSynchronize();
    memcpy(date,num,size);
    hipFree(num);
    

    return 0;
}
