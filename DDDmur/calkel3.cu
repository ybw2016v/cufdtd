
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

extern "C"
{
    int cal(float *P1,float *VX,float *VY,float *VZ,float *Z0,int xar,int yar,int zar,int xm,int ym,int zm,int n,int n2,float m,int flag);
}


__global__ void cul1(float * p1j,float * vx1j,float * vy1j,float * vz1j,float *z0j,int xr, int yr,int zr,float m)
{
    // 计算kel1，用于进行梯度运算。
    // printf("%f \n",m);
    int i,j,k;
    i=threadIdx.x;
    j=blockIdx.x;
    k=blockIdx.y;

    if ((i>=1)&&(i<=blockDim.x-2)&&(j>=1)&&(j<=gridDim.x-2)&&(k>=1)&&(k<=gridDim.y-2))
    {
    vx1j[j*yr+i*xr+k*zr]-=(p1j[j*yr+i*xr+k*zr]-p1j[j*yr+(i-1)*xr+k*zr])/z0j[j*yr+i*xr+k*zr]/m;
    vy1j[j*yr+i*xr+k*zr]-=(p1j[(j+1)*yr+i*xr+k*zr]-p1j[j*yr+(i)*xr+k*zr])/z0j[j*yr+i*xr+k*zr]/m;
    vz1j[j*yr+i*xr+k*zr]-=(p1j[(j)*yr+i*xr+(k+1)*zr]-p1j[j*yr+(i)*xr+k*zr])/z0j[j*yr+i*xr+k*zr]/m;
    }
    else if (i==0)
    {
        p1j[(j)*yr+i*xr+(k)*zr]=p1j[(j)*yr+(i+2)*xr+(k)*zr];
    }

    else if (i==blockDim.x-1)
    {
        p1j[(j)*yr+i*xr+(k)*zr]=p1j[(j)*yr+(i-2)*xr+(k)*zr];
    }

    else if (j==0)
    {
        p1j[(j)*yr+i*xr+(k)*zr]=p1j[(j+2)*yr+(i)*xr+(k)*zr];
    }

    else if (j==gridDim.x-1)
    {
        p1j[(j)*yr+i*xr+(k)*zr]=p1j[(j-2)*yr+(i)*xr+(k)*zr];
    }

    else if (k==0)
    {
        p1j[(j)*yr+i*xr+(k)*zr]=p1j[(j)*yr+(i)*xr+(k+2)*zr];
    }

    else if (k==gridDim.y-1)
    {
        p1j[(j)*yr+i*xr+(k)*zr]=p1j[(j)*yr+(i)*xr+(k-2)*zr];
    }

    

}


__global__ void cul2(float * p1j,float * vx1j,float * vy1j,float * vz1j,float *z0j,int xr, int yr,int zr,float m) 
{
    // 计算kel2，用于计算压强。
    int i,j,k;
    i=threadIdx.x;
    j=blockIdx.x;
    k=blockIdx.y;
    if((i>=1)&&(i<=blockDim.x-2)&&(j>=1)&&(j<=gridDim.x-2)&&(k>=1)&&(k<=gridDim.y-2))
    {
        if(i==1)
        {
            float dog;
            dog=p1j[j*yr+(i-1)*xr+k*zr]-(vx1j[j*yr+(i+2)*xr+k*zr]-vx1j[j*yr+(i+1)*xr+k*zr]+vy1j[j*yr+(i+1)*xr+k*zr]-vy1j[(j-1)*yr+(i+1)*xr+k*zr]+vz1j[j*yr+(i+1)*xr+k*zr]-vz1j[(j)*yr+(i+1)*xr+(k-1)*zr])*z0j[j*yr+(i+1)*xr+k*zr]/m;
            p1j[j*yr+i*xr+k*zr]=p1j[j*yr+(i-1)*xr+k*zr]+(1.0-m)/(1.0+m)*(dog-p1j[j*yr+i*xr+k*zr]);
        }
        else if (i==blockDim.x-2)
        {
            float dog;
            dog=p1j[j*yr+(i+1)*xr+k*zr]-(vx1j[j*yr+(i)*xr+k*zr]-vx1j[j*yr+(i-1)*xr+k*zr]+vy1j[j*yr+(i-1)*xr+k*zr]-vy1j[(j-1)*yr+(i-1)*xr+k*zr]+vz1j[j*yr+(i-1)*xr+k*zr]-vz1j[(j)*yr+(i-1)*xr+(k-1)*zr])*z0j[j*yr+(i-1)*xr+k*zr]/m;
            p1j[j*yr+i*xr+k*zr]=p1j[j*yr+(i+1)*xr+k*zr]+(1.0-m)/(1.0+m)*(dog-p1j[j*yr+i*xr+k*zr]);
        }
        else if (j==1)
        {
            float dog;
            dog=p1j[(j-1)*yr+i*xr+k*zr]-(vx1j[(j+1)*yr+(i+1)*xr+k*zr]-vx1j[(+1)*yr+i*xr+k*zr]+vy1j[(j+1)*yr+i*xr+k*zr]-vy1j[(j)*yr+i*xr+k*zr]+vz1j[(j+1)*yr+i*xr+k*zr]-vz1j[(j+1)*yr+i*xr+(k-1)*zr])*z0j[(j+1)*yr+i*xr+k*zr]/m;
            p1j[j*yr+i*xr+k*zr]=p1j[(j-1)*yr+i*xr+k*zr]+(1.0-m)/(1.0+m)*(dog-p1j[j*yr+i*xr+k*zr]);
        }
        else if (j==gridDim.x-2)
        {
            float dog;
            dog=p1j[(j+1)*yr+i*xr+k*zr]-(vx1j[(j-1)*yr+(i+1)*xr+k*zr]-vx1j[(j-1)*yr+i*xr+k*zr]+vy1j[(j-1)*yr+i*xr+k*zr]-vy1j[(j-2)*yr+i*xr+k*zr]+vz1j[(j-1)*yr+i*xr+k*zr]-vz1j[(j-1)*yr+i*xr+(k-1)*zr])*z0j[(j-1)*yr+i*xr+k*zr]/m;
            p1j[j*yr+i*xr+k*zr]=p1j[(j+1)*yr+i*xr+k*zr]+(1.0-m)/(1.0+m)*(dog-p1j[j*yr+i*xr+k*zr]);
        }
        else if (k==1)
        {
            float dog;
            dog=p1j[j*yr+i*xr+(k-1)*zr]-(vx1j[j*yr+(i+1)*xr+(k+1)*zr]-vx1j[j*yr+i*xr+(k+1)*zr]+vy1j[j*yr+i*xr+(k+1)*zr]-vy1j[(j-1)*yr+i*xr+(k+1)*zr]+vz1j[j*yr+i*xr+(k+1)*zr]-vz1j[(j)*yr+i*xr+(k)*zr])*z0j[j*yr+i*xr+(k+1)*zr]/m;
            p1j[j*yr+i*xr+k*zr]=p1j[(j-1)*yr+i*xr+k*zr]+(1.0-m)/(1.0+m)*(dog-p1j[j*yr+i*xr+k*zr]);
        }
        else if(k==gridDim.y-2)
        {
            float dog;
            dog=p1j[j*yr+i*xr+(k+1)*zr]-(vx1j[j*yr+(i+1)*xr+(k-1)*zr]-vx1j[j*yr+i*xr+(k-1)*zr]+vy1j[j*yr+i*xr+(k-1)*zr]-vy1j[(j-1)*yr+i*xr+(k-1)*zr]+vz1j[j*yr+i*xr+(k-1)*zr]-vz1j[(j)*yr+i*xr+(k-2)*zr])*z0j[j*yr+i*xr+(k-1)*zr]/m;
            p1j[j*yr+i*xr+k*zr]=p1j[(j+1)*yr+i*xr+k*zr]+(1.0-m)/(1.0+m)*(dog-p1j[j*yr+i*xr+k*zr]);
        }


        else if ((i>=2)&&(i<=blockDim.x-3)&&(j>=2)&&(j<=gridDim.x-3)&&(k>=2)&&(k<=gridDim.y-3))
        {
        p1j[j*yr+i*xr+k*zr]-=(vx1j[j*yr+(i+1)*xr+k*zr]-vx1j[j*yr+i*xr+k*zr]+vy1j[j*yr+i*xr+k*zr]-vy1j[(j-1)*yr+i*xr+k*zr]+vz1j[j*yr+i*xr+k*zr]-vz1j[(j)*yr+i*xr+(k-1)*zr])*z0j[j*yr+i*xr+k*zr]/m;
        }
    }
}

int cal(float *P1,float *VX,float *VY,float *VZ,float *Z0,int xar,int yar,int zar,int xm,int ym,int zm,int n,int n2,float m,int flag)
{
    // printf("%f %f %f\n",m,P1[400],Z0[400]);
    int size;
    float * p1=NULL;
    float * vx=NULL;
    float * vy=NULL;
    float * vz=NULL;
    float * z0=NULL;
    
    xar=xar/sizeof(float);
    yar=yar/sizeof(float);
    zar=zar/sizeof(float);
    size=(zm*zar)*sizeof(float);
    // printf("%d %d \n",xar,yar);
    // printf("%d %d %d %d\n",xar,yar,xm,ym);
    // printf("%d \n",size);
    // cudaMallocManaged((void**)&p1, size);
    
    hipMallocManaged((void**)&p1, size);
    hipMallocManaged((void**)&vx, size);
    hipMallocManaged((void**)&vy, size);
    hipMallocManaged((void**)&vz, size);
    hipMallocManaged((void**)&z0, size);
    hipDeviceSynchronize();
    // printf("%hd \n",p1);
    memcpy(p1,P1,size);
    
    memcpy(p1,P1,size);
    memcpy(vx,VX,size);
    memcpy(vz,VZ,size);
    memcpy(vy,VY,size);
    memcpy(z0,Z0,size);
    // printf("%s \n", "OK");
    hipDeviceSynchronize();
    for (int i = n; i < n2; i++)
    {
        p1[128*xar+128*yar+128*zar]=sin(0.008*i);
        dim3 dog(ym,zm);
        cul1<<<dog,xm>>>(p1,vx,vy,vz,z0,xar,yar,zar,m);
        hipDeviceSynchronize();
        cul2<<<dog,xm>>>(p1,vx,vy,vz,z0,xar,yar,zar,m);
        hipDeviceSynchronize();
    }
    hipDeviceSynchronize();
    // for(int i = 0; i<ym; i++)
    // {
        
    //     for(int j = 0; j <xm; j++)
    //     {
    //         printf("%f ",Z0[j*xar+i*yar]);
    //     }
    //     printf("\n");
        
    // }
    memcpy(P1,p1,size);
    memcpy(VX,vx,size);
    memcpy(VZ,vz,size);
    memcpy(VY,vy,size);
    hipFree(p1);
    hipFree(vx);
    hipFree(vz);
    hipFree(vy);
    hipFree(z0);
    
    return 0;
}
