
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

extern "C"
{
    int cal(float *P1,float *VX,float *VZ,float *Z0,float *LX,float *LZ,int xar,int yar,int xm,int ym,int n,int n2,float m,int flag);
}


__global__ void cul1(float * p1j,float * vx1j,float * vz1j,float *z0j,float *lx,float *ly,int xr, int yr,float m)
{
    // 计算kel1，用于进行梯度运算。
    // printf("%f \n",m);
    int i,j;
    i=threadIdx.x;
    j=blockIdx.x;
    if((i>=1)&&(j<gridDim.x-1))
    {
        vx1j[j*yr+i*xr]-=(p1j[j*yr+i*xr]-p1j[j*yr+(i-1)*xr])/z0j[j*yr+i*xr]/m;
        vz1j[j*yr+i*xr]-=(p1j[(j+1)*yr+i*xr]-p1j[j*yr+i*xr])/z0j[j*yr+i*xr]/m;
    }
    if(j==1)
    {
        lx[i]=p1j[(j)*yr+i*xr];
    }
    if(j==gridDim.x-2)
    {
        lx[blockDim.x+i]=p1j[(j)*yr+i*xr];
    }
    if (i==1)
    {
        ly[j]=p1j[(j)*yr+i*xr];
    }
    if (i==blockDim.x-2)
    {
        ly[gridDim.x+j]=p1j[(j)*yr+i*xr];
    }

}


__global__ void cul2(float * p1j,float * vx1j,float * vz1j,float *z0j,float *lx,float *ly,int xr, int yr,float m) 
{
    // 计算kel2，用于计算压强。
    int i,j;
    i=threadIdx.x;
    j=blockIdx.x;
    if (j==0)
    {
        float dog;
        dog=lx[i]-(vx1j[(j+2)*yr+i*xr]-vx1j[(j+1)*yr+i*xr]+vz1j[(j+1)*yr+i*xr]-vz1j[(j+1)*yr+(i-1)*xr])*z0j[j*yr+i*xr]/m;
        p1j[j*yr+i*xr]=lx[i]+(1.0-m)/(1.0+m)*(dog-p1j[j*yr+i*xr]);    
    }

    else if(j==gridDim.x-1)
    {
        float dog;
        dog=lx[blockDim.x+i]-(vx1j[(j)*yr+i*xr]-vx1j[(j-1)*yr+i*xr]+vz1j[(j-1)*yr+i*xr]-vz1j[(j-1)*yr+(i-1)*xr])*z0j[j*yr+i*xr]/m;
        p1j[j*yr+i*xr]=lx[blockDim.x+i]+(1.0-m)/(1.0+m)*(dog-p1j[j*yr+i*xr]); 
    }
    else if (i==0)
    {
        float dog;
        dog=ly[i]-(vx1j[(j+1)*yr+(i+1)*xr]-vx1j[(j)*yr+(i+1)*xr]+vz1j[(j)*yr+(i+1)*xr]-vz1j[(j)*yr+(i)*xr])*z0j[j*yr+i*xr]/m;
        p1j[j*yr+i*xr]=ly[i]+(1.0-m)/(1.0+m)*(dog-p1j[j*yr+i*xr]);    

    }
    else if (i==blockDim.x-1)
    {
        float dog;
        dog=lx[gridDim.x+j]-(vx1j[(j+1)*yr+(i-1)*xr]-vx1j[(j)*yr+(i-1)*xr]+vz1j[(j)*yr+(i-1)*xr]-vz1j[(j)*yr+(i-2)*xr])*z0j[j*yr+i*xr]/m;
        p1j[j*yr+i*xr]=lx[blockDim.x+j]+(1.0-m)/(1.0+m)*(dog-p1j[j*yr+i*xr]);
    }
    else
    {
    p1j[j*yr+i*xr]-=(vx1j[j*yr+(i+1)*xr]-vx1j[j*yr+i*xr]+vz1j[j*yr+i*xr]-vz1j[(j-1)*yr+i*xr])*z0j[j*yr+i*xr]/m;
    }

}

int cal(float *P1,float *VX,float *VZ,float *Z0,float *LX,float *LZ,int xar,int yar,int xm,int ym,int n,int n2,float m,int flag)
{
    // printf("%f %f %f\n",m,P1[400],Z0[400]);
    int size;
    float * p1=NULL;
    float * vx=NULL;
    float * vz=NULL;
    float * z0=NULL;
    float * lx=NULL;
    float * lz=NULL;

    
    xar=xar/sizeof(float);
    yar=yar/sizeof(float);
    size=(ym*yar)*sizeof(float);
    // printf("%d %d \n",xar,yar);
    // printf("%d %d %d %d\n",xar,yar,xm,ym);
    // printf("%d \n",size);
    // cudaMallocManaged((void**)&p1, size);
    
    hipMallocManaged((void**)&p1, size);
    hipMallocManaged((void**)&vx, size);
    hipMallocManaged((void**)&vz, size);
    hipMallocManaged((void**)&z0, size);
    hipMallocManaged((void**)&lx, 2*xm*sizeof(float));
    hipMallocManaged((void**)&lz, 2*ym*sizeof(float));
    hipDeviceSynchronize();
    // printf("%hd \n",p1);
    memcpy(p1,P1,size);
    
    memcpy(p1,P1,size);
    memcpy(vx,VX,size);
    memcpy(vz,VZ,size);
    memcpy(z0,Z0,size);
    memcpy(lx,LX,2*xm*sizeof(float));
    memcpy(lz,LZ,2*ym*sizeof(float));
    // printf("%s \n", "OK");
    hipDeviceSynchronize();
    for (int i = n; i < n2; i++)
    {
        p1[200*xar+200*yar]=sin(0.008*i);
        cul1<<<ym-2,xm-2>>>(p1,vx,vz,z0,lx,lz,xar,yar,m);
        hipDeviceSynchronize();
        cul2<<<ym-2,xm-2>>>(p1,vx,vz,z0,lx,lz,xar,yar,m);
        hipDeviceSynchronize();
    }
    hipDeviceSynchronize();
    // for(int i = 0; i<ym; i++)
    // {
        
    //     for(int j = 0; j <xm; j++)
    //     {
    //         printf("%f ",Z0[j*xar+i*yar]);
    //     }
    //     printf("\n");
        
    // }
    memcpy(P1,p1,size);
    memcpy(VX,vx,size);
    memcpy(VZ,vz,size);

    hipFree(p1);
    hipFree(vx);
    hipFree(vz);
    hipFree(z0);
    
    return 0;
}

